#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void calcolo_oracolo(float *a, float *b, float *oracolo, int n);

int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int M;
    float* h_a = 0;     // Host array a
    float* d_a;         // Device array a
    float* h_b = 0;     // Host array b
    float *d_b;         // Device array b
    float result = 0;   // Risultato finale
    float oracolo = 0.0;

    hipEvent_t start, stop; // eventi per il calcolo del tempo di esecuzione
    float elapsed_time = 0.0;
    float tempo_sequenziale = 0.0;
	
    printf("Inserisci lunghezza vettori: ");
    scanf("%d", &M);

    h_a = (float *)malloc (M * sizeof (*h_a));      // Alloco h_a e lo inizializzo
    if (!h_a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    
    h_b = (float *)malloc (M * sizeof (*h_b));  // Alloco h_b e lo inizializzo
    if (!h_b) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }

    srand(time(NULL));
    for (int i = 0; i < M; i++) {
        h_a[i] = ((float)rand() * 4 / (float)RAND_MAX) - 2; //reali nell'intervallo (-2,+2)
        h_b[i] = ((float)rand() * 4 / (float)RAND_MAX) - 2;
    }

    /* Calcolo Oracolo e tempo di esecuzione sequenziale */
    clock_t inizio = clock();    
    calcolo_oracolo(h_a, h_b, &oracolo, M);
    clock_t fine = clock();
    tempo_sequenziale = (float)(fine - inizio) / CLOCKS_PER_SEC;    

    cudaStat = hipMalloc ((void**)&d_a, M*sizeof(*h_a));       // Alloco d_a
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    cudaStat = hipMalloc ((void**)&d_b, M*sizeof(*h_b));       // Alloco d_b
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    stat = hipblasCreate(&handle);               // Creo l'handle per cublas
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetVector(M,sizeof(float),h_a,1,d_a,1);    // Setto h_a su d_a
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_a);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetVector(M,sizeof(float),h_b,1,d_b,1);    // Setto h_b su d_b
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    /* Creiamo gli eventi per il tempo */
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); // tempo di inizio

    stat = hipblasSdot(handle,M,d_a,1,d_b,1,&result);        // Calcolo il prodotto
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed hipblasSdot");
        hipFree (d_a);
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    hipEventRecord(stop); // tempo di fine

    /* Calcolo tempo di esecuzione */
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    
    printf("Risultato del prodotto --> %f\n",result);
    printf("Oracolo: %f\n", oracolo);
    printf("Tempo di esecuzione cublas: %fs\n", elapsed_time/1000);
    printf("Tempo di esecuzione sequenziale: %fs\n", tempo_sequenziale);
    printf("Speedup: %f\n", tempo_sequenziale/(elapsed_time/1000));

    /* Rilascio degli eventi */
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipFree (d_a);     // Dealloco d_a
    hipFree (d_b);     // Dealloco d_b
    
    hipblasDestroy(handle);  // Distruggo l'handle
    
    free(h_a);      // Dealloco h_a
    free(h_b);      // Dealloco h_b    
    return EXIT_SUCCESS;
}

void calcolo_oracolo(float *a, float *b, float *oracolo, int n) {
    for (int i = 0; i < n; i++) {
        *oracolo += a[i] * b[i];
    }
}
