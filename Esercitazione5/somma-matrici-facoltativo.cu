#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void sommaMatrixGPU(double *a, double *b, double *s, int m);
void sommmaMatrixSequenziale(double *a, double *b, double *oracolo, int n, int m); 

int main(void) {
    double *a_host, *b_host, *s_host, *oracolo;
    double *a_dev, *b_dev, *s_dev;
    float elapsed_time = 0.0, tempo_sequenziale = 0.0;
    int n, m;

    hipEvent_t start, stop;

    /* Input: dimensione della matrice */
    printf("Inserire dimensionioni delle matrici: ");
    scanf("%d %d", &n, &m);

    /* Allocazione memoria host */
    a_host = (double*) malloc((n * m) * sizeof(double));
    b_host = (double*) malloc((n * m) * sizeof(double));
    s_host = (double*) malloc((n * m) * sizeof(double));
    oracolo = (double*) malloc((n * m) * sizeof(double));

    /* Inizializzazione pseudocasuale delle matrici */
    srand(time(NULL));
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            a_host[i * m + j] = ((double)rand() * 4 / (double)RAND_MAX) - 2;
            b_host[i * m + j] = ((double)rand() * 4 / (double)RAND_MAX) - 2;
        }
    }

    /* Calcolo Oracolo e Tempo d'esecuzione Sequenziale */
    clock_t inizio = clock();    
    sommmaMatrixSequenziale(a_host, b_host, oracolo, n, m);
    clock_t fine = clock();
    tempo_sequenziale = (float)(fine - inizio) / CLOCKS_PER_SEC; 

    /* Allocazione memoria device */
    hipMalloc((void **) &a_dev, (n * m) * sizeof(double));
    hipMalloc((void **) &b_dev, (n * m) * sizeof(double));
    hipMalloc((void **) &s_dev, (n * m) * sizeof(double));

    /* Copia dei vettori da host a device */
    hipMemcpy(a_dev, a_host, (n * m) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b_host, (n * m) * sizeof(double), hipMemcpyHostToDevice);

    /* Configurazione del Kernel */
    dim3 blockDim(8, 8); // (8, 8) ottimale
    dim3 gridDim(
        (n + blockDim.x - 1) / blockDim.x,
        (m + blockDim.y - 1) / blockDim.y
    );
    printf("blockDim = (%d,%d)\n", blockDim.x, blockDim.y);
    printf("gridDim = (%d,%d)\n", gridDim.x, gridDim.y);

    /* Creiamo gli eventi per il tempo */
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start); // tempo di inizio

    /* Inovcazione del Kernel */
    sommaMatrixGPU<<<gridDim, blockDim>>>(a_dev, b_dev, s_dev, m);

    /* Calcolo tempo di esecuzione */
    hipEventRecord(stop); // tempo di fine
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);

    /* Copia del risultato da device ad host */
    hipMemcpy(s_host, s_dev, (n * m) * sizeof(double), hipMemcpyDeviceToHost);

    /* Stampa dei risultati */
    if (n <= 5 && m <= 5) {
        printf("\nMatrice A:\n");
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < m; j++) {
                printf("%f\t", a_host[i * m + j]);
            }
            printf("\n");
        }
        printf("\nMatrice B:\n");
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < m; j++) {
                printf("%f\t", b_host[i * m + j]);
            }
            printf("\n");
        }
        printf("\nMatrice Somma:\n");
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < m; j++) {
                printf("%f\t", s_host[i * m + j]);
            }
            printf("\n");
        }
        printf("\nOracolo:\n");
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < m; j++) {
                printf("%f\t", s_host[i * m + j]);
            }
            printf("\n");
        }
    }
    printf("\nTempo di esecuzione parallelo (GPU): %fs\n", elapsed_time/1000);
    printf("\nTempo di esecuzione sequenziale (CPU): %fs\n", tempo_sequenziale);

    /* Rilascio degli eventi */
    hipEventDestroy(start);
    hipEventDestroy(stop);

    /* free della memoria */
    free(a_host);
    free(b_host);
    free(s_host);
    free(oracolo);
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(s_dev);

    return 0;
}

__global__ void sommaMatrixGPU(double *a, double *b, double *s, int m) {
    int i = threadIdx.x + (blockDim.x * blockIdx.x);
    int j = threadIdx.y + (blockDim.y * blockIdx.y);
    s[i * m + j] = a[i * m + j] + b[i * m + j];
}

void sommmaMatrixSequenziale(double *a, double *b, double *oracolo, int n, int m) {
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            oracolo[i * m + j] = a[i * m + j] + b[i * m + j];
        }
    }
} 


